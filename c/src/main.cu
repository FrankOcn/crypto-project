#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <limits.h>
#include <gmp.h>
#include <cudam.cuh>

#define CHECK(call) \
{ \
const hipError_t error = call; \
if (error != hipSuccess) \
{ \
printf("Error: %s:%d, ", __FILE__, __LINE__); \
printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
exit(1); \
} \
}

#define MOD_ELEM_0 135291589527
#define MOD_ELEM_1 18446744073705357312
#define MOD_ELEM_2 9223371487099224063
#define MOD_ELEM_3 512


struct vec_t {
  uint32_t * elems;
  int size;
  int capacity;
};

void insert(struct vec_t* vec, uint32_t elem) {
  if (vec->size == vec->capacity) {
    vec->capacity = vec->capacity * 2;
    uint32_t* newElems = (uint32_t *)malloc(vec->capacity* sizeof(uint32_t));
    for (int i = 0; i < vec->size; ++i) {
      newElems[i] = vec->elems[i];
    }
    free(vec->elems);
    vec->elems = newElems;
  }
  vec->elems[vec->size] = elem;
  vec->size++;
}

struct vec_t* initVec() {
  struct vec_t* newVec = (struct vec_t*)malloc(sizeof(struct vec_t));
  newVec->capacity = 64;
  newVec->size = 0;
  newVec->elems = (uint32_t *)malloc(newVec->capacity * 2 * sizeof(uint32_t));
  return newVec;
}

__global__ void checkFactor(uint32_t* primes, uint32_t* results, cudam_t comp) {
  uint32_t prime = primes[(threadIdx.x) + blockIdx.x * blockDim.x];
  uint32_t* result = results + (threadIdx.x + blockIdx.x * blockDim.x);

  cudam_t c = cudam_init();
  cudam_t p = cudam_init();

  p[0] = prime;
  while (cudam_is_zero(c) == 1 && cudam_cmp(p, comp) != 1) {
    cudam_set(comp, c);
    cudam_mod(c, c, p);
    if (cudam_is_zero(c) == 1) {
      cudam_mult_ui(p, p, prime);
      *result = *result + 1;
    }
  }
  free(c);
  free(p);
}

int main(int argc, char ** argv) {

  cudam_t modulus = cudam_init();
  modulus[0] = MOD_ELEM_0;
  modulus[1] = MOD_ELEM_1;
  modulus[2] = MOD_ELEM_2;
  modulus[3] = MOD_ELEM_3;

  printf("Loading prime factor base\n");

  FILE *fp;

  mpz_t mQr, mNr, mP, mQrTest, mPTest;
  mpz_init(mQr);
  mpz_init(mNr);
  mpz_init(mP);
  mpz_init(mQrTest);
  mpz_init(mPTest);
  mpz_set_ui(mQr, 1);
  mpz_set_ui(mNr, 1);
  mpz_import(mP, 4, -1, sizeof(uint64_t), 0, 0, modulus);
  mpz_set(mQrTest, mP);
  mpz_add_ui(mQrTest, mQrTest, 1);
  mpz_divexact_ui(mQrTest, mQrTest, 4);

  fp = fopen("./primes.txt", "r");
  vec_t * primeVector = initVec();
  int c = 0;
  uint32_t a;
  while (c != -1) {
    a = 0;
    do {
      c = fgetc(fp);
      if (c >= '0' && c <= '9') {
        a *= 10;
        uint32_t n = c - '0';
        a += n;
      }
    } while (c >= '0' && c <= '9');
    if (a != 0) {
      mpz_set_ui(mPTest, a);
      mpz_powm(mPTest, mPTest, mQrTest, mP);
      mpz_powm_ui(mPTest, mPTest, 2, mP);
      if (mpz_cmp_d(mPTest, a) == 0) {
        mpz_mul_ui(mQr, mQr, a);
        insert(primeVector, a);
      } else {
        mpz_mul_ui(mNr, mNr, a);
      }
    }
  }

  mpz_clear(mQrTest);
  mpz_clear(mPTest);
  fclose(fp);


  gmp_printf("mQr: %Zd\n", mQr);
  gmp_printf("mNr: %Zd\n", mNr);

  dim3 block (1024);
  dim3 grid (((primeVector->size)+block.x - 1)/block.x);

  printf("Done loading prime factor base\n");

  printf("Moving factor base into GPU\n");

  uint32_t * dPrimes;

  hipMalloc((uint32_t**)&dPrimes, primeVector->size * sizeof(uint32_t));
  hipMemcpy(dPrimes, primeVector->elems, primeVector->size * sizeof(uint32_t) , hipMemcpyHostToDevice);

  printf("Creating results array\n");

  uint32_t* results = (uint32_t*) malloc(primeVector->size * sizeof(uint32_t));

  uint32_t * dResults;

  hipMalloc((uint32_t**)&dResults, primeVector->size * sizeof(uint32_t));
  cudam_t five = cudam_init();
  five[0] = 5;
  cudam_t testNum = cudam_init();
  testNum[0] = 1;


  // Let's move it along a bit.
  int pow = 1;
  for (; pow < 87; pow++) {
    cudam_multm(testNum, testNum, five, modulus);
    printf("power%d: %llu,%llu,%llu,%llu\n", pow, testNum[0], testNum[1], testNum[2], testNum[3]);
  }

  cudam_t max = cudam_init();
  max[4] = 1;

  uint64_t* testNumGPU;

  hipMalloc((uint64_t**)&testNumGPU, 4* sizeof(uint64_t));

  cudam_t candidate = cudam_init();
  cudam_t fiveTest = cudam_init();

  while (1) {

    cudam_multm(testNum, testNum, testNum, modulus);

    pow += 1;

    printf("pow: %d\n", pow);

    cudam_mod(fiveTest, testNum, five);
    mpz_t mTest, mGcd;
    mpz_init(mTest);
    mpz_init(mGcd);
    mpz_import(mTest, 4, -1, sizeof(uint64_t), 0, 0, testNum);

    mpz_gcd(mGcd, mTest, mNr);


    if (mpz_cmp_ui(mGcd, 1) != 0) {
      //printf("Has a factor in mNr. Continuing.\n");
      mpz_clear(mTest);
      mpz_clear(mGcd);
      continue;
    }

    mpz_gcd(mGcd, mTest, mQr);
    int hasNonQFactor = 0;
    while (hasNonQFactor == 0 && mpz_cmp(mTest, mGcd) != 0) {
      if (mpz_cmp_ui(mGcd, 1) == 0) {
        //printf("Has a factor not in mQr. Continuing.\n");
        hasNonQFactor = 1;
        continue;
      }
      mpz_divexact(mTest, mTest, mGcd);
      mpz_gcd(mGcd, mTest, mQr);
    }

    mpz_clear(mTest);
    mpz_clear(mGcd);

    if (hasNonQFactor == 1) continue;

    // Reset results array
    for (int i = 0; i < primeVector->size; ++i) {
      results[i] = 0;
    }

    //printf("Move result array into GPU\n");

    hipMemcpy(dResults, results, primeVector->size * sizeof(uint32_t), hipMemcpyHostToDevice);

    //printf("testNum: %llu, %llu, %llu, %llu\n", testNum[0], testNum[1], testNum[2], testNum[3]);

    hipMemcpy(testNumGPU, testNum, 4 * sizeof(uint64_t), hipMemcpyHostToDevice);

    //printf("about to run\n");

    checkFactor <<< block, grid >>>(dPrimes, dResults, testNumGPU);
    CHECK(hipDeviceSynchronize());
    hipMemcpy(results, dResults, primeVector->size * sizeof(uint32_t), hipMemcpyDeviceToHost);

    //printf("results copied back\n");

    cudam_zero(candidate);
    candidate[0] = 1;

    for (int i = 0; i < primeVector->size; ++i) {
      if (results[i] != 0) {
        printf("result: %d prime: %d\n", results[i], primeVector->elems[i]);
        for (int j = 0; j < results[i]; ++j) {
          cudam_mult_ui(candidate, candidate, primeVector->elems[i]);
        }
      }
    }

    for (int i = 0; i < 4; ++i) {
      printf("cand: %llu test: %llu\n", candidate[i], testNum[i]);
    }
    if (cudam_cmp(candidate, testNum) == 0) {
      printf("FOUND OMG FOUND ONE OMG OMG OMG %d\n", pow);

      for (int i = 0; i < 4; ++i) {
        printf("cand: %llu test: %llu\n", candidate[i], testNum[i]);
      }
      return 0;
      mpz_clear(mQr);
      mpz_clear(mNr);
      mpz_clear(mP);
    }
  }

}
